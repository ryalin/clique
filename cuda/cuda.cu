#include "hip/hip_runtime.h"
#include <map>
#include <set>
#include <hip/hip_runtime.h>
#include <>

#define BLOCKDIMX 32
#define BLOCKDIMY 32
#define THREADCOUNT 1024

__global__ void kernelCliqueSolver() {

}

bool parallelCuda(std::map<int,std::set<int>> graph, int targetCount) {
  dim3 blockDim(BLOCKDIMX, BLOCKDIMY);
  dim3 gridDim((image->height + BLOCKDIMX - 1) / BLOCKDIMX,
                (image->width + BLOCKDIMY - 1) / BLOCKDIMY);

    float* device_x;
    float* device_y;
    float* device_result;

    // allocate device memory buffers on the GPU using hipMalloc
    hipMalloc(&device_x, sizeof(float) * N);
    hipMalloc(&device_y, sizeof(float) * N);
    hipMalloc(&device_result, sizeof(float) * N);

    // start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();

    // copy input arrays to the GPU using hipMemcpy
    hipMemcpy(device_x, xarray, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(device_y, yarray, sizeof(float) * N, hipMemcpyHostToDevice);

  kernelRenderCircles<<<gridDim, blockDim>>>();
  hipDeviceSynchronize();
  return true;
}